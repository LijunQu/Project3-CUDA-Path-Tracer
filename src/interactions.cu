#include "hip/hip_runtime.h"
#include "interactions.h"

#include "utilities.h"

#include <thrust/random.h>

__host__ __device__ glm::vec3 calculateRandomDirectionInHemisphere(
    glm::vec3 normal,
    thrust::default_random_engine &rng)
{
    thrust::uniform_real_distribution<float> u01(0, 1);

    float up = sqrt(u01(rng)); // cos(theta)
    float over = sqrt(1 - up * up); // sin(theta)
    float around = u01(rng) * TWO_PI;

    // Find a direction that is not the normal based off of whether or not the
    // normal's components are all equal to sqrt(1/3) or whether or not at
    // least one component is less than sqrt(1/3). Learned this trick from
    // Peter Kutz.

    glm::vec3 directionNotNormal;
    if (abs(normal.x) < SQRT_OF_ONE_THIRD)
    {
        directionNotNormal = glm::vec3(1, 0, 0);
    }
    else if (abs(normal.y) < SQRT_OF_ONE_THIRD)
    {
        directionNotNormal = glm::vec3(0, 1, 0);
    }
    else
    {
        directionNotNormal = glm::vec3(0, 0, 1);
    }

    // Use not-normal direction to generate two perpendicular directions
    glm::vec3 perpendicularDirection1 =
        glm::normalize(glm::cross(normal, directionNotNormal));
    glm::vec3 perpendicularDirection2 =
        glm::normalize(glm::cross(normal, perpendicularDirection1));

    return up * normal
        + cos(around) * over * perpendicularDirection1
        + sin(around) * over * perpendicularDirection2;
}


__host__ __device__ glm::vec3 squareToDiskConcentric(const glm::vec2& xi) {
    float a = 2.0f * xi.x - 1.0f;
    float b = 2.0f * xi.y - 1.0f;

    float r, phi;
    if (a > -b) {
        if (a > b) { r = a; phi = (PI * 0.25f) * (b / a); }
        else { r = b; phi = (PI * 0.25f) * (2.0f - (a / b)); }
    }
    else {
        if (a < b) { r = -a; phi = (PI * 0.25f) * (4.0f + (b / a)); }
        else { r = -b; phi = (b != 0.0f) ? (PI * 0.25f) * (6.0f - (a / b)) : 0.0f; }
    }

    float u = r * cosf(phi);
    float v = r * sinf(phi);
    return glm::vec3(u, v, 0.0f);
}

// Map square -> cosine-weighted hemisphere (local +Z is the normal)
__host__ __device__ glm::vec3 squareToHemisphereCosine(const glm::vec2& xi) {
    glm::vec3 d = squareToDiskConcentric(xi);
    float x = d.x, y = d.y;
    float z = sqrtf(fmaxf(0.0f, 1.0f - x * x - y * y));
    return glm::normalize(glm::vec3(x, y, z));
}

// Build an ONB for a given normal
__host__ __device__ inline void buildOnb(const glm::vec3& n, glm::vec3& t, glm::vec3& b) {
    // choose a helper vector not parallel to n
    glm::vec3 h = (fabsf(n.x) > 0.1f) ? glm::vec3(0, 1, 0) : glm::vec3(1, 0, 0);
    b = glm::normalize(glm::cross(n, h));
    t = glm::cross(b, n);
}

// World-space cosine hemisphere sample w/ matching pdf
__host__ __device__ glm::vec3 cosineSampleHemisphere(const glm::vec3& n,
    thrust::default_random_engine& rng) {
    thrust::uniform_real_distribution<float> u01(0.f, 1.f);
    glm::vec2 xi(u01(rng), u01(rng));            // two uniform randoms in [0,1)
    glm::vec3 local = squareToHemisphereCosine(xi);

    // rotate local (+Z up) into world frame defined by n
    glm::vec3 t, b; buildOnb(n, t, b);
    glm::vec3 wi = glm::normalize(local.x * t + local.y * b + local.z * n);

    float cosTheta = fmaxf(0.f, glm::dot(n, wi));
    return wi;
}

__host__ __device__ void bsdf_pdf(PathSegment& pathSegment,
    glm::vec3 normal, float& pdf)
{
    float cosTheta = glm::max(0.f, glm::dot(pathSegment.ray.direction, normal));
    pdf = cosTheta / PI;
}


__host__ __device__ void bsdf_specular(PathSegment& pathSegment,
    glm::vec3 intersect,
    glm::vec3 normal,
    float& pdf,
    const Material& m,
    thrust::default_random_engine& rng)
{

    glm::vec3 dir = pathSegment.ray.direction;

    pathSegment.ray.origin = intersect + EPSILON * normal;
    pathSegment.ray.direction = glm::normalize(glm::reflect(dir, normal));
    float cosTheta = glm::max(0.f, glm::dot(pathSegment.ray.direction, normal));
    bsdf_pdf(pathSegment, normal, pdf);
    pathSegment.color *= m.color;
}


__host__ __device__ float schlick(float cos, float reflectIdx)
{
    float r0 = powf((1.0f - reflectIdx) / (1.0f + reflectIdx), 2.0f);
    return r0 + (1.0f - r0) * powf((1.0f - cos), 5.0f);
}

__host__ __device__ void bsdf_diffuse(PathSegment& pathSegment,
    glm::vec3 intersect,
    glm::vec3 normal,
    float& pdf,
    const Material& m,
    thrust::default_random_engine& rng)
{

    glm::vec3 dir = calculateRandomDirectionInHemisphere(normal, rng);
    pathSegment.ray.origin = intersect + EPSILON * normal;
    //pathSegment.ray.direction = dir;
    pathSegment.ray.direction = dir;

    bsdf_pdf(pathSegment, normal, pdf);
    pathSegment.color *= m.color;
}


__host__ __device__ void btdf_specular(PathSegment& pathSegment,
    glm::vec3 intersect,
    glm::vec3 normal,
    float& pdf,
    const Material& m,
    thrust::default_random_engine& rng)
{
    //float etaA = 1.;
    //float etaB = 1.55;
    float eta;
    glm::vec3 n;
    glm::vec3 rayDir = pathSegment.ray.direction;
    if (glm::dot(normal, rayDir) <= 0.f) {
        //eta = etaB / etaA;
        eta = m.indexOfRefraction;
        n = normal;
    }
    else {
        //eta = etaA / etaB;
        eta = 1.0f / m.indexOfRefraction;
        n = -normal;
    }
    glm::vec3 wt = glm::refract(rayDir, n, eta);

    // TIR
    if (glm::length(wt) < 0.01f) {
        pathSegment.color *= 0.0f;
        wt = glm::reflect(rayDir, n);
    }

    float cos = glm::clamp(glm::dot(rayDir, n), 0.0f, 1.0f);
    float reflectProb = schlick(cos, m.indexOfRefraction);

    thrust::uniform_real_distribution<float> u01(0, 1);
    float rand = u01(rng);

    pathSegment.ray.direction = reflectProb < rand ? glm::normalize(glm::reflect(rayDir, n)) : glm::normalize(wt);
    pathSegment.ray.origin = intersect + 0.01f * pathSegment.ray.direction;

    pathSegment.color *= m.color;



}

__host__ __device__ float FresnelDielectricEval(float cosThetaI, float IOR) {
    float etaI = 1.f;
    float etaT = IOR;
    cosThetaI = glm::clamp(cosThetaI, -1.f, 1.f);

    if (cosThetaI > 0.f) {
        float temp = etaI;
        etaI = etaT;
        etaT = temp;
    }
    cosThetaI = glm::abs(cosThetaI);

    float sinThetaI = glm::sqrt(glm::max(0.f, 1.f - cosThetaI * cosThetaI));
    float sinThetaT = etaI / etaT * sinThetaI;
    float cosThetaT = glm::sqrt(glm::max(0.f, 1.f - sinThetaT * sinThetaT));
    float Rparl = ((etaT * cosThetaI) - (etaI * cosThetaT)) /
        ((etaT * cosThetaI) + (etaI * cosThetaT));
    float Rperp = ((etaI * cosThetaI) - (etaT * cosThetaT)) /
        ((etaI * cosThetaI) + (etaT * cosThetaT));

    return (Rparl * Rparl + Rperp * Rperp) * 0.5f;
}

__host__ __device__ void glass(PathSegment& pathSegment,
    glm::vec3 intersect,
    glm::vec3 normal,
    float& pdf,
    const Material& m,
    thrust::default_random_engine& rng)
{
    thrust::uniform_real_distribution<float> u01(0, 1);
    float rand = u01(rng);

    float cos = glm::dot(pathSegment.ray.direction, normal);
    float fresnel = FresnelDielectricEval(cos, m.indexOfRefraction);

    if (rand < fresnel) {
        bsdf_specular(pathSegment, intersect, normal, pdf, m, rng);
    }
    else {
        btdf_specular(pathSegment, intersect, normal, pdf, m, rng);
    }
    
}



__host__ __device__ void bsdf(PathSegment& pathSegment,
    glm::vec3 intersect,
    glm::vec3 normal,
    float& pdf,
    const Material& m,
    thrust::default_random_engine& rng)
{
    if (m.hasReflective && m.hasRefractive) {
        glass(pathSegment, intersect, normal, pdf, m, rng);
    }
    else if (m.hasReflective) {
        bsdf_specular(pathSegment, intersect, normal, pdf, m, rng);
    }
    else if (m.hasRefractive) {
        btdf_specular(pathSegment, intersect, normal, pdf, m, rng);
    }
    else {
        bsdf_diffuse(pathSegment, intersect, normal, pdf, m, rng);
    }

}





__host__ __device__ void scatterRay(
    PathSegment& pathSegment,
    glm::vec3 intersect,
    glm::vec3 normal,
    const Material& m,
    thrust::default_random_engine &rng)
{
    // TODO: implement this.
    // A basic implementation of pure-diffuse shading will just call the
    // calculateRandomDirectionInHemisphere defined above.

    /*
    glm::vec3 dir = calculateRandomDirectionInHemisphere(normal, rng);

    pathSegment.ray.origin = intersect + EPSILON * normal;
    //pathSegment.ray.direction = dir;
    pathSegment.ray.direction = glm::normalize(dir);


    pathSegment.color *= m.color;
    */
    float pdf;
    bsdf(pathSegment, intersect, normal, pdf, m, rng);

}
